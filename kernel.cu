#include "hip/hip_runtime.h"
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "hip/hip_vector_types.h"
#include "configs.h"
#include "kernel.cuh"

typedef unsigned int uint;

// simulation parameters in constant memory
__constant__ struct Configuration params;

struct integrate_functor
{
    float deltaTime;

    __host__ __device__
        integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __device__
        void operator()(Tuple t)
    {
        volatile float2 posData = thrust::get<0>(t);
        volatile float2 velData = thrust::get<1>(t);
        float2 pos = make_float2(posData.x, posData.y);
        float2 vel = make_float2(velData.x, velData.y);

        pos += vel * deltaTime;

        if (dot(pos, pos) >= params.m_radius * params.m_radius) {
            float SQRT = (dot(pos, vel) / dot(vel, vel)) * (dot(pos, vel) / dot(vel, vel)) + ((params.m_radius * params.m_radius - dot(pos, pos)) / dot(vel, vel));
            float dt1 = fabs(sqrt(fabs(SQRT)) - dot(pos, vel) / dot(vel, vel));
            
            float2 pos_1 = pos + vel * dt1;
            float2 vel_1 = vel;

            float2 nx = -pos_1 / params.m_radius;

            float vel_n = dot(vel_1, nx);

            float2 vel_1_n = vel_1 - 2 * nx * vel_n;

            vel = vel_1_n;

            pos = pos_1 + vel * dt1;
        }

        // store new position and velocity
        thrust::get<0>(t) = pos;
        thrust::get<1>(t) = vel;
    }
};
void integrateSystem(float* pos, float* vel, float deltaTime, uint numParticles)
{
    thrust::device_ptr<float2> d_pos2((float2*)pos);
    thrust::device_ptr<float2> d_vel2((float2*)vel);
    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2, d_vel2)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2 + numParticles, d_vel2 + numParticles)),
        integrate_functor(deltaTime));
}