#include "hip/hip_runtime.h"
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "hip/hip_vector_types.h"
#include "configs.h"
#include "kernel.cuh"

typedef unsigned int uint;

// simulation parameters in constant memory
__constant__ struct Configuration params;

struct integrate_functor
{
    float deltaTime;

    __host__ __device__
        integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __device__
        void operator()(Tuple t)
    {
        volatile float2 posData = thrust::get<0>(t);
        volatile float2 velData = thrust::get<1>(t);
        float2 pos = make_float2(posData.x, posData.y);
        float2 vel = make_float2(velData.x, velData.y);

        pos += vel * deltaTime;

        if (dot(pos, pos) >= params.m_radius * params.m_radius) {
            float SQRT = (dot(pos, vel) / dot(vel, vel)) * (dot(pos, vel) / dot(vel, vel)) + ((params.m_radius * params.m_radius - dot(pos, pos)) / dot(vel, vel));
            float dtPart = fabs(sqrt(fabs(SQRT)) - dot(pos, vel) / dot(vel, vel));
            
            float2 posTMP = pos + vel * dtPart;
            float2 velTMP = vel;

            float2 normal = -posTMP / params.m_radius;

            float velNormal = dot(velTMP, normal);

            float2 velPart = velTMP - 2 * normal * velNormal;

            vel = velPart;

            pos = posTMP + vel * dtPart;
        }

        // store new position and velocity
        thrust::get<0>(t) = pos;
        thrust::get<1>(t) = vel;
    }
};
void integrateSystem(float* pos, float* vel, float deltaTime, uint numParticles)
{
    thrust::device_ptr<float2> d_pos2((float2*)pos);
    thrust::device_ptr<float2> d_vel2((float2*)vel);
    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2, d_vel2)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2 + numParticles, d_vel2 + numParticles)),
        integrate_functor(deltaTime));
}