#include "hip/hip_runtime.h"

#include "cuda_func.h"
#include <iostream>
#define CUDA_LAUNCH_BLOCKING 1
__global__ void test_kernel(int* first, int* second, const size_t size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        first[index] += second[index];
}
namespace CUDA {
    void helloWorld(void) {
        const size_t size = 45;
        int first[size];
        int second[size];
        for (int i = 0; i < size; i++) {
            first[i] = i + 5;
            second[i] = i - 2;
        }
        const size_t mem_size = sizeof(first);
        int *d_first;
        hipMalloc(&d_first, mem_size);
        int *d_second;
        hipMalloc(&d_second, mem_size);

        hipMemcpy(d_first, first, mem_size, hipMemcpyHostToDevice);
        hipMemcpy(d_second, second, mem_size, hipMemcpyHostToDevice);


        test_kernel <<<6, 9>>> (d_first, d_second, size);

        hipMemcpy(first, d_first, mem_size, hipMemcpyDeviceToHost);

        for (int i = 0; i < size; i++)
            std::cout << first[i] << " ";
        std::cout << std::endl;
    }
}