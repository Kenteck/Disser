#include "hip/hip_runtime.h"
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"
#include "hip/hip_vector_types.h"
#include "configs.h"
#include "kernel.cuh"
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

typedef unsigned int uint;

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)
static const char* _cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const* const func, const char* const file,
    int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

inline void __getLastCudaError(const char* errorMessage, const char* file,
    const int line) {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err) {
        fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d) %s.\n",
            file, line, errorMessage, static_cast<int>(err),
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// simulation parameters in constant memory
__device__ __constant__ struct Configuration params;

struct integrate_functor
{
    float deltaTime;

    __host__ __device__
        integrate_functor(float delta_time) : deltaTime(delta_time) {}

    template <typename Tuple>
    __device__
        void operator()(Tuple t)
    {
        volatile float2 posData = thrust::get<0>(t);
        volatile float2 velData = thrust::get<1>(t);
        float2 pos = make_float2(posData.x, posData.y);
        float2 vel = make_float2(velData.x, velData.y);

        pos += vel * deltaTime;

        if (dot(pos, pos) >= params.m_radius * params.m_radius) {
            float SQRT = (dot(pos, vel) / dot(vel, vel)) * (dot(pos, vel) / dot(vel, vel)) + ((params.m_radius * params.m_radius - dot(pos, pos)) / dot(vel, vel));
            float dtPart = fabs(sqrt(fabs(SQRT)) - dot(pos, vel) / dot(vel, vel));
            
            float2 posTMP = pos + vel * dtPart;
            float2 velTMP = vel;
            float HypotVelocity = length(vel);

            float2 normal = -posTMP / params.m_radius;

            float velNormal = dot(velTMP, normal);

            float2 velPart = velTMP - 2 * normal * velNormal;

            vel = velPart;

            pos = posTMP + vel * dtPart;
            
            vel = (vel / length(vel)) * HypotVelocity; // comepnsation of the float multiplication
        }

        // store new position and velocity
        
        thrust::get<0>(t) = pos;
        thrust::get<1>(t) = vel;
    }
};

void integrateSystem(float* pos, float* vel, float deltaTime, uint numParticles)
{
    thrust::device_ptr<float2> d_pos2((float2*)pos);
    thrust::device_ptr<float2> d_vel2((float2*)vel);
    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2, d_vel2)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos2 + numParticles, d_vel2 + numParticles)),
        integrate_functor(deltaTime));
}

struct momentum_functor
{
    template <typename Tuple>
    __device__
        void operator()(Tuple t)
    {
        volatile float2 posData = thrust::get<0>(t);
        volatile float2 velData = thrust::get<1>(t);
        volatile float momentumData = thrust::get<2>(t);
        
        float2 pos = make_float2(posData.x, posData.y);
        float2 vel = make_float2(velData.x, velData.y);

        float momentum = pos.x * vel.y - pos.y * vel.x;

        thrust::get<2>(t) = momentum;
    }
};

float integrateMomentumOfSystem(float* pos, float* vel, float* momentum, uint numParticles)
{
    thrust::device_ptr<float2> d_pos((float2*)pos);
    thrust::device_ptr<float2> d_vel((float2*)vel);
    thrust::device_ptr<float> d_momentum((float*)momentum);
    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_vel, d_momentum)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos + numParticles, d_vel + numParticles, d_momentum + numParticles)),
        momentum_functor());
    return thrust::reduce(d_momentum, d_momentum + numParticles, 0);;
}


//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

// calculate position in uniform grid
__device__ int2 calcGridPos(float2 p)
{
    int2 gridPos;
    gridPos.x = floor((p.x) / params.m_sizeOfCell);
    gridPos.y = floor((p.y) / params.m_sizeOfCell);
    return gridPos;
}

// calculate address in grid from position (clamping to edges)
__device__ uint calcGridHash(int2 gridPos)
{
    gridPos.x = gridPos.x & (params.m_gridSize - 1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.m_gridSize - 1);
    return __umul24(gridPos.y, params.m_gridSize) + gridPos.x;
}


// calculate grid hash value for each particle
__global__ void calcHashD(uint* gridParticleHash,  // output
    uint* gridParticleIndex, // output
    float2* pos,               // input: positions
    uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= numParticles) return;

    volatile float2 p = pos[index];

    // get address in grid
    int2 gridPos = calcGridPos(make_float2(p.x, p.y));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

void calcHash(uint* gridParticleHash,
    uint* gridParticleIndex,
    float* pos,
    uint    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 1024, numBlocks, numThreads);
    
    // execute the kernel
    calcHashD << < numBlocks, numThreads >> > (gridParticleHash,
        gridParticleIndex,
        (float2*)pos,
        numParticles);
    
    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
}

void sortParticles(uint* dGridParticleHash, uint* dGridParticleIndex, uint numParticles)
{
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
        thrust::device_ptr<uint>(dGridParticleHash + numParticles),
        thrust::device_ptr<uint>(dGridParticleIndex));
}

// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
__global__
void reorderDataAndFindCellStartD(uint* cellStart,        // output: cell start index
    uint* cellEnd,          // output: cell end index
    float2* sortedPos,        // output: sorted positions
    float2* sortedVel,        // output: sorted velocities
    uint* gridParticleHash, // input: sorted grid hashes
    uint* gridParticleIndex,// input: sorted particle indices
    float2* oldPos,           // input: sorted position array
    float2* oldVel,           // input: sorted velocity array
    uint    numParticles)
{
    // Handle to thread block group
    
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    uint hash;

    // handle case when no. of particles not multiple of block size
    if (index < numParticles)
    {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x + 1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[index - 1];
        }
    }

    cg::sync(cta);

    if (index < numParticles)
    {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x])
        {
            cellStart[hash] = index;

            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1)
        {
            cellEnd[hash] = index + 1;
        }

        // Now use the sorted index to reorder the pos and vel data
        uint sortedIndex = gridParticleIndex[index];
        float2 pos = oldPos[sortedIndex];
        float2 vel = oldVel[sortedIndex];

        sortedPos[index] = pos;
        sortedVel[index] = vel;
    }


}

void reorderDataAndFindCellStart(uint* cellStart,
    uint* cellEnd,
    float* sortedPos,
    float* sortedVel,
    uint* gridParticleHash,
    uint* gridParticleIndex,
    float* oldPos,
    float* oldVel,
    uint   numParticles,
    uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 1024, numBlocks, numThreads);

    // set all cells to empty
    checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)));

    uint smemSize = sizeof(uint) * (numThreads + 1);
    reorderDataAndFindCellStartD << < numBlocks, numThreads, smemSize >> > (
        cellStart,
        cellEnd,
        (float2*)sortedPos,
        (float2*)sortedVel,
        gridParticleHash,
        gridParticleIndex,
        (float2*)oldPos,
        (float2*)oldVel,
        numParticles);
    getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

}

// collide a particle against all other particles in a given cell
__device__
float2 collideCell(int2 gridPos,
    uint    index,
    float2* oldPos,
    float2* oldVel,
    uint* cellStart,
    uint* cellEnd)
{
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = cellStart[gridHash];

    float2 force = make_float2(0.0f, 0.0f);

    if (startIndex != 0xffffffff)          // cell is not empty
    {
        // iterate over particles in this cell
        uint endIndex = cellEnd[gridHash];

        for (uint j = startIndex; j < endIndex; j++)
        {
            if (index != j) {
                float dist = length(oldPos[j] - oldPos[index]);
                if (dist <= params.m_interactionDistance) {
                    float Hypot = length(oldVel[j]);
                    force += oldVel[j] / Hypot;
                }
            }
        }
    }

    return force;
}

__global__
void collideD(float2* newVel,               // output: new velocity
    float2* oldPos,               // input: sorted positions
    float2* oldVel,               // input: sorted velocities
    uint* gridParticleIndex,    // input: sorted particle indices
    uint* cellStart,
    uint* cellEnd,
    uint  numParticles,
    hiprandState* states)
{
    uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (index >= numParticles) return;

    hiprand_init(index, index, 0, &states[index]);   // 	Initialize CURAND

    // read particle data from sorted arrays
    float2 pos = oldPos[index];
    float2 vel = oldVel[index];

    // get address in grid
    int2 gridPos = calcGridPos(pos);

    // examine neighbouring cells
    float2 force = make_float2(0.0f, 0.0f);

    for (int y = -1; y <= 1; y++)
    {
        for (int x = -1; x <= 1; x++)
        {
            int2 neighbourPos = gridPos + make_int2(x, y);
            force += collideCell(neighbourPos, index, oldPos, oldVel, cellStart, cellEnd);
        }
    }

    if (length(force) > 1e-5) {
        force.x += params.m_noice * (1 - 2 * (hiprand_normal(&states[index])));
        force.y += params.m_noice * (1 - 2 * (hiprand_normal(&states[index])));
        float HypotForce = length(force);

        float2 Ort = force / HypotForce;

        float HypotVelocity = length(vel);

        vel = Ort * HypotVelocity;
    }
    else 
    {
        float HypotVelocity = length(vel);

        vel.x += params.m_noice * (1 - 2 * (hiprand_normal(&states[index])));
        vel.y += params.m_noice * (1 - 2 * (hiprand_normal(&states[index])));

        float HypotForce = length(vel);

        float2 Ort = vel / HypotForce;

        vel = Ort * HypotVelocity;
    }
    // write new velocity back to original unsorted location
    uint originalIndex = gridParticleIndex[index];
    newVel[originalIndex] = vel;
}


void collide(float* newVel,
    float* sortedPos,
    float* sortedVel,
    uint* gridParticleIndex,
    uint* cellStart,
    uint* cellEnd,
    uint   numParticles,
    uint   numCells,
    hiprandState* states)
{

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    collideD << < numBlocks, numThreads >> > ((float2*)newVel,
        (float2*)sortedPos,
        (float2*)sortedVel,
        gridParticleIndex,
        cellStart,
        cellEnd,
        numParticles,
        states);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");

}

void setParameters(Configuration* hostParams)
{
    // copy parameters to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(Configuration)));
}