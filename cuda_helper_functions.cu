#include <windows.h>
#include "cuda_func.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "configs.h"

static const char* _cudaGetErrorEnum(hipError_t error) {
    return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const* const func, const char* const file,
    int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

// simulation parameters in constant memory
__constant__ Configuration params;

typedef unsigned int uint;

extern "C"
{
    void cudaInit()
    {
        int devID = 0;
        int device_count = 0;
        printf("CUDA initizalition: started\n");

        checkCudaErrors(hipGetDeviceCount(&device_count));
        printf("CUDA devices found: %i\n", device_count);

        if (device_count == 0) {
            fprintf(stderr,
                "gpuGetMaxGflopsDeviceId() CUDA error:"
                " no devices supporting CUDA.\n");
        }

        checkCudaErrors(hipSetDevice(devID)); // Set the firts available device
        printf("CUDA initizalition: finished\n");
    }

    void allocateArray(void** devPtr, size_t size)
    {
        checkCudaErrors(hipMalloc(devPtr, size));
    }

    void freeArray(void* devPtr)
    {
        checkCudaErrors(hipFree(devPtr));
    }

    void threadSync()
    {
        checkCudaErrors(hipDeviceSynchronize());
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size)
    {
        checkCudaErrors(hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice));
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource** cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone));
    }

    void unregisterGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
    }

    void* mapGLBufferObject(struct hipGraphicsResource** cuda_vbo_resource)
    {
        void* ptr;
        checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes,
            *cuda_vbo_resource));
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
    }
}